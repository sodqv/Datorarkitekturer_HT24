
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
__global__ void multKernel(int n, float* a, float* b, float* c)
{
	for (int i = 0; i < n; i++) {
		c[i] = a[i] * b[i];
	}
}

int main() {
	int N = 1<<24;
	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;
	
	// Allocate host memory
	h_a = new float[N];
	h_b = new float[N];
	h_c = new float[N];

	// Allocate device memory
	hipMalloc(&d_a, N * sizeof(float));
	hipMalloc(&d_b, N * sizeof(float));
	hipMalloc(&d_c, N * sizeof(float));

	// Initialize host data
	for (int i = 0; i < N; i++)
	{
		h_a[i] = 2.0f;
		h_b[i] = 3.0f;
	}

	// Copy data from host to device
	hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

	// Launch the kernel
	multKernel<<<1, 1>>>(N, d_a, d_b, d_c);
	
	// Copy result back to host
	hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

	// Check result for errors (all values should be 6.0f)

	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(h_c[i] - 6.0f));

	std::cout << "Max error: " << maxError << std::endl;
	// Clean up

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	delete[] h_a;
	delete[] h_b;
	delete[] h_c;
	
	return 0;
}

